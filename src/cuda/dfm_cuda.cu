#include "hip/hip_runtime.h"
// Maintainer: enrico-lattuada

/*! \file dfm_cuda.cu
    \brief Definition of core CUDA Digital Fourier Microscopy functions
*/

// *** headers ***
#include "dfm_cuda.cuh"

#include "helper_debug.cuh"
#include "helper_cufft.cuh"
#include "helper_dfm_cuda.cuh"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <stdlib.h>

// #include <chrono>
// using namespace std::chrono;

#define CUFFTCOMPLEX hipfftDoubleComplex

// *** code ***

/*!
    Evaluate the device memory pitch for multiple subarrays of size N with 8bytes elements
*/
void cudaGetDevicePitch8B(size_t N,
                          size_t &pitch)
{
    double *d_arr;

    gpuErrchk(hipMallocPitch(&d_arr, &pitch, N * sizeof(double), 2));

    pitch /= sizeof(double);

    gpuErrchk(hipFree(d_arr));
}

/*!
    Evaluate the device memory pitch for multiple subarrays of size N with 4bytes elements
*/
void cudaGetDevicePitch4B(size_t N,
                          size_t &pitch)
{
    float *d_arr;

    gpuErrchk(hipMallocPitch(&d_arr, &pitch, N * sizeof(float), 2));

    pitch /= sizeof(float);

    gpuErrchk(hipFree(d_arr));
}

/*!
    Evaluate the device memory pitch for multiple subarrays of size N with 2bytes elements
*/
void cudaGetDevicePitch2B(size_t N,
                          size_t &pitch)
{
    int16_t *d_arr;

    gpuErrchk(hipMallocPitch(&d_arr, &pitch, N * sizeof(int16_t), 2));

    pitch /= sizeof(int16_t);

    gpuErrchk(hipFree(d_arr));
}

/*!
    Evaluate the device memory pitch for multiple subarrays of size N with 2bytes elements
*/
void cudaGetDevicePitch1B(size_t N,
                          size_t &pitch)
{
    int8_t *d_arr;

    gpuErrchk(hipMallocPitch(&d_arr, &pitch, N * sizeof(int8_t), 2));

    pitch /= sizeof(int8_t);

    gpuErrchk(hipFree(d_arr));
}

/*!
    Evaluate the device memory size in bytes for fft2
*/
void cudaGetFft2MemSize(size_t nx,
                        size_t ny,
                        size_t batch,
                        size_t *memsize)
{
    fft2_get_mem_size(nx,
                      ny,
                      batch,
                      memsize);
}

/*!
    Evaluate the device memory size in bytes for fft
*/
void cudaGetFftMemSize(size_t nt,
                       size_t batch,
                       size_t pitch,
                       size_t *memsize)
{
    fft_get_mem_size(nt,
                     batch,
                     pitch,
                     memsize);
}

/*!
    Transfer images on GPU and compute fft2
 */
template <typename T>
void compute_fft2(const T *h_in,
                  double *h_out,
                  size_t width,
                  size_t height,
                  size_t length,
                  size_t nx,
                  size_t ny,
                  size_t num_fft2,
                  size_t buff_pitch)
{
    // compute half width of fft2
    size_t _nx = nx / 2 + 1;
    // compute batch number of fft2
    size_t batch = (length - 1) / num_fft2 + 1;
    // compute fft2 normalizaton factor
    double norm_fact = 1.0 / sqrt((double)(nx * ny));

    // ***Allocate device arrays
    // workspace
    double *d_workspace;
    gpuErrchk(hipMalloc(&d_workspace, 2 * _nx * ny * batch * sizeof(double)));
    // buffer (only allocate if T is not double)
    T *d_buff;
    if (!std::is_same<T, double>::value)
    {
        gpuErrchk(hipMalloc(&d_buff, buff_pitch * height * batch * sizeof(T)));
    }

    // ***Create fft2 plan
    hipfftHandle fft2_plan = fft2_create_plan(nx,
                                             ny,
                                             batch);

    // Compute efficient execution configuration
    // copy/convert kernel
    int blockSize_copy = 512;                                                           // The launch configurator returned block size
    int gridSize_copy = (width * height * batch + blockSize_copy - 1) / blockSize_copy; // The actual grid size needed, based on input size

    // scale kernel
    int blockSize_scale; // The launch configurator returned block size
    int minGridSize;     // The minimum grid size needed to achieve the
                         // maximum occupancy for a full device launch
    int gridSize_scale;  // The actual grid size needed, based on input size

    gpuErrchk(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize_scale, scale_array_kernel, 0, 0));
    // Round up according to array size
    gridSize_scale = (2 * _nx * ny * batch + blockSize_scale - 1) / blockSize_scale;

    // ***Batched fft2
    for (size_t ii = 0; ii < num_fft2; ii++)
    {
        // rezero workspace array
        gpuErrchk(hipMemset(d_workspace, 0.0, 2 * _nx * ny * batch * sizeof(double)));

        // ***Copy values to device
        if (std::is_same<T, double>::value)
        {
            // copy values directly to workspace with zero padding
            // number of images to copy
            size_t num_imgs_copy = (ii + 1) * batch > length ? length - ii * batch : batch;
            // use hipMemcpy3D
            hipMemcpy3DParms params = {0};
            params.srcArray = NULL;
            params.srcPos = make_hipPos(0, 0, ii * batch);
            params.srcPtr = make_hipPitchedPtr((double *)h_in, width * sizeof(double), width, height);
            params.dstArray = NULL;
            params.dstPos = make_hipPos(0, 0, 0);
            params.dstPtr = make_hipPitchedPtr(d_workspace, 2 * _nx * sizeof(double), 2 * _nx, ny);
            params.extent = make_hipExtent(width * sizeof(double), height, num_imgs_copy);
            params.kind = hipMemcpyHostToDevice;

            gpuErrchk(hipMemcpy3D(&params));
        }
        else
        {
            // rezero buffer array
            gpuErrchk(hipMemset(d_buff, (T)0, buff_pitch * height * batch * sizeof(T)));

            // offset index
            size_t offset = ii * width * height * batch;
            // number of rows to copy
            size_t num_rows_copy = (ii + 1) * batch > length ? height * (length - ii * batch) : height * batch;
            // copy values to buffer
            gpuErrchk(hipMemcpy2D(d_buff, buff_pitch * sizeof(T), h_in + offset, width * sizeof(T), width * sizeof(T), num_rows_copy, hipMemcpyHostToDevice));

            // convert values of buffer into workspace
            copy_convert_kernel<<<gridSize_copy, blockSize_copy>>>(d_buff,
                                                                   d_workspace,
                                                                   width,
                                                                   width * height,
                                                                   buff_pitch,
                                                                   buff_pitch * height,
                                                                   2 * _nx,
                                                                   2 * _nx * ny,
                                                                   width * height * batch);
        }

        // ***Execute fft2 plan
        cufftSafeCall(hipfftExecD2Z(fft2_plan, d_workspace, (CUFFTCOMPLEX *)d_workspace));

        // ***Normalize fft2
        // Starting index
        size_t start = 2 * ii * _nx * ny * batch;
        // Final index (if exceeds array size, truncate)
        size_t end = (ii + 1) * batch > length ? 2 * length * _nx * ny : 2 * (ii + 1) * _nx * ny * batch;
        // scale array
        scale_array_kernel<<<gridSize_scale, blockSize_scale>>>(d_workspace,
                                                                norm_fact,
                                                                d_workspace,
                                                                end - start);
                                                                
        // ***Copy values back to host
        gpuErrchk(hipMemcpy(h_out + start, d_workspace, (end - start) * sizeof(double), hipMemcpyDeviceToHost));
    }

    // ***Free memory
    gpuErrchk(hipFree(d_workspace));
    gpuErrchk(hipFree(d_buff));
    cufftSafeCall(hipfftDestroy(fft2_plan));
}

template void compute_fft2<double>(const double *h_in, double *h_out, size_t width, size_t height, size_t length, size_t nx, size_t ny, size_t num_fft2, size_t buff_pitch);
template void compute_fft2<float>(const float *h_in, double *h_out, size_t width, size_t height, size_t length, size_t nx, size_t ny, size_t num_fft2, size_t buff_pitch);
template void compute_fft2<int64_t>(const int64_t *h_in, double *h_out, size_t width, size_t height, size_t length, size_t nx, size_t ny, size_t num_fft2, size_t buff_pitch);
template void compute_fft2<int32_t>(const int32_t *h_in, double *h_out, size_t width, size_t height, size_t length, size_t nx, size_t ny, size_t num_fft2, size_t buff_pitch);
template void compute_fft2<int16_t>(const int16_t *h_in, double *h_out, size_t width, size_t height, size_t length, size_t nx, size_t ny, size_t num_fft2, size_t buff_pitch);
template void compute_fft2<u_int64_t>(const u_int64_t *h_in, double *h_out, size_t width, size_t height, size_t length, size_t nx, size_t ny, size_t num_fft2, size_t buff_pitch);
template void compute_fft2<u_int32_t>(const u_int32_t *h_in, double *h_out, size_t width, size_t height, size_t length, size_t nx, size_t ny, size_t num_fft2, size_t buff_pitch);
template void compute_fft2<u_int16_t>(const u_int16_t *h_in, double *h_out, size_t width, size_t height, size_t length, size_t nx, size_t ny, size_t num_fft2, size_t buff_pitch);
template void compute_fft2<u_int8_t>(const u_int8_t *h_in, double *h_out, size_t width, size_t height, size_t length, size_t nx, size_t ny, size_t num_fft2, size_t buff_pitch);
