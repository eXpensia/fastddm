#include "hip/hip_runtime.h"
// Maintainer: enrico-lattuada

/*! \file helper_prefix_sum.cu
    \brief Definition of helper functions for cumulative sum (prefix sum) on GPU
*/

// *** headers ***
#include "helper_prefix_sum.cuh"
#include "helper_debug.cuh"

#include <hip/hip_runtime.h>

// definitions
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
//#ifdef ZERO_BANK_CONFLICTS
//#define CONFLICT_FREE_OFFSET(n) \ 
// ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
//#else
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
//#endif

unsigned long int BLOCK_SIZE = 512;
unsigned long int ELEMENTS_PER_BLOCK = BLOCK_SIZE * 2;

// *** code ***

// See:
// https://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/scan/doc/scan.pdf
// https://github.com/lxxue/prefix_sum/blob/master/prefix_sum.cu

/*!
    Scan multiple large arrays on the GPU
 */
void scanManyLargeArrays(double *output,
                         double *input,
                         unsigned long int length,
                         unsigned long int dist,
                         unsigned long int N)
{
    unsigned long int Nx = length / ELEMENTS_PER_BLOCK;             // number of even blocks per row
    unsigned long int remainder = length - Nx * ELEMENTS_PER_BLOCK; // remainder from even blocks

    if (remainder == 0)
    {
        scanManyLargeEvenArrays(output,
                                input,
                                length,
                                dist,
                                Nx,
                                N);
    }
    else
    {
        unsigned long int length_even = length - remainder;

        // copy the last element of the even part of the subarray before prefix sum
        double *a1;
        gpuErrchk(hipMalloc(&a1, N * sizeof(double)));
        int numSMs;
        gpuErrchk(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0));
        int gridSize_copy = min((size_t)N, 32 * (size_t)numSMs);
        copy_every_kernel<<<gridSize_copy, 1>>>(a1,
                                                input + length_even - 1,
                                                dist,
                                                N);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // scan over even arrays
        scanManyLargeEvenArrays(output,
                                input,
                                length_even,
                                dist,
                                Nx,
                                N);

        // scan the remaining elements
        scanManySmallArrays(output + length_even,
                            input + length_even,
                            remainder,
                            dist,
                            N);

        // copy the last element of the even part of the subarray after prefix sum
        double *a2;
        gpuErrchk(hipMalloc(&a2, N * sizeof(double)));
        copy_every_kernel<<<gridSize_copy, 1>>>(a2,
                                                output + length_even - 1,
                                                dist,
                                                N);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // sum the last element of the even arrays portion
        // and the first element of the remainder input
        // to the remainder scan
        add_many_kernel<<<gridSize_copy, remainder>>>(output+length_even,
                                                      dist,
                                                      Nx,
                                                      Nx * N,
                                                      remainder,
                                                      a1,
                                                      a2);
        gpuErrchk(hipPeekAtLastError());

        // free memory
        gpuErrchk(hipFree(a1));
        gpuErrchk(hipFree(a2));
    }
}

/*!
    Scan multiple large even arrays on the GPU
 */
void scanManyLargeEvenArrays(double *output,
                             double *input,
                             unsigned long int length,
                             unsigned long int dist,
                             unsigned long int Nx,
                             unsigned long int N)
{
    // compute execution parameters
    const int shared_mem_size = ELEMENTS_PER_BLOCK * sizeof(double);
    int numSMs;
    gpuErrchk(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0));
    int gridSize = min((size_t)Nx * (size_t)N, 32 * (size_t)numSMs);

    // allocate partial sums and incr arrays
    double *sums, *incr;
    gpuErrchk(hipMalloc(&sums, Nx * N * sizeof(double)));
    gpuErrchk(hipMalloc(&incr, Nx * N * sizeof(double)));

    // do scan
    prescan_many_even_kernel<<<gridSize, BLOCK_SIZE, 2 * shared_mem_size>>>(output,
                                                                            input,
                                                                            dist,
                                                                            Nx,
                                                                            Nx * N,
                                                                            ELEMENTS_PER_BLOCK,
                                                                            sums);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    // compute increment
    unsigned long int scan_sum_threads_needed = (Nx + 1) / 2;
    if (scan_sum_threads_needed > BLOCK_SIZE)
    {
        scanManyLargeArrays(incr,
                            sums,
                            Nx,
                            Nx,
                            N);
    }
    else
    {
        scanManySmallArrays(incr,
                            sums,
                            Nx,
                            Nx,
                            N);
    }

    // add increment
    add_many_kernel<<<gridSize, ELEMENTS_PER_BLOCK>>>(output,
                                                      dist,
                                                      Nx,
                                                      Nx * N,
                                                      ELEMENTS_PER_BLOCK,
                                                      incr);
    gpuErrchk(hipPeekAtLastError());

    // free memory
    gpuErrchk(hipFree(sums));
    gpuErrchk(hipFree(incr));
}

/*!
    Scan multiple small arrays on the GPU
 */
void scanManySmallArrays(double *output,
                         double *input,
                         unsigned long int length,
                         unsigned long int dist,
                         unsigned long int N)
{
    unsigned long int powerOfTwo = nextPowerOfTwo(length);

    // Compute execution parameters
    const int shared_mem_size = powerOfTwo * sizeof(double);
    int numSMs;
    gpuErrchk(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0));
    int gridSize = min((size_t)N, 32 * (size_t)numSMs);
    int blockSize = (length + 1) / 2;

    // do scan
    prescan_many_arbitrary_kernel<<<gridSize, blockSize, 2 * shared_mem_size>>>(output,
                                                                                input,
                                                                                dist,
                                                                                N,
                                                                                length,
                                                                                powerOfTwo);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
}

/*!
    Wrapper function to compute cumulative sum on multiple subarrays on the GPU
 */
void scan_wrap(double *output,
               double *input,
               unsigned long int length,
               unsigned long int dist,
               unsigned long int N)
{
    if (length > ELEMENTS_PER_BLOCK)
    {
        scanManyLargeArrays(output,
                            input,
                            length,
                            dist,
                            N);
    }
    else
    {
        scanManySmallArrays(output,
                            input,
                            length,
                            dist,
                            N);
    }
}

/*!
    Step 1 of cumsummany:
    Compute cumulative sum on multiple even portions of the subarrays on the GPU
 */
__global__ void prescan_many_even_kernel(double *output,
                                         double *input,
                                         unsigned long int dist,
                                         unsigned long int Nx,
                                         unsigned long int NxNy,
                                         unsigned long int n,
                                         double *sums)
{
    extern __shared__ double temp[];

    for (unsigned long int blockID = blockIdx.x; blockID < NxNy; blockID += gridDim.x)
    {
        unsigned long int threadID = threadIdx.x;
        unsigned long int y = (blockID / Nx);
        unsigned long int blockOffset = y * dist + (blockID - y * Nx) * n;

        // load input into shared memory
        unsigned long int ai = threadID;
        unsigned long int bi = threadID + (n / 2);
        unsigned long int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
        unsigned long int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
        temp[ai + bankOffsetA] = input[blockOffset + ai];
        temp[bi + bankOffsetB] = input[blockOffset + bi];

        // build sum in place up the tree
        unsigned long int offset = 1;
        for (unsigned long int d = n >> 1; d > 0; d >>= 1)
        {
            __syncthreads();
            if (threadID < d)
            {
                unsigned long int ai = offset * (2 * threadID + 1) - 1;
                unsigned long int bi = offset * (2 * threadID + 2) - 1;
                ai += CONFLICT_FREE_OFFSET(ai);
                bi += CONFLICT_FREE_OFFSET(bi);

                temp[bi] += temp[ai];
            }
            offset *= 2;
        }
        __syncthreads();

        // write total sum to sums and clear the last element
        if (threadID == 0)
        {
            sums[blockID] = temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)];
            temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0.0;
        }

        // traverse down tree and build scan
        for (unsigned long int d = 1; d < n; d *= 2)
        {
            offset >>= 1;
            __syncthreads();
            if (threadID < d)
            {
                unsigned long int ai = offset * (2 * threadID + 1) - 1;
                unsigned long int bi = offset * (2 * threadID + 2) - 1;
                ai += CONFLICT_FREE_OFFSET(ai);
                bi += CONFLICT_FREE_OFFSET(bi);

                double t = temp[ai];
                temp[ai] = temp[bi];
                temp[bi] += t;
            }
        }
        __syncthreads();

        output[blockOffset + ai] = temp[ai + bankOffsetA];
        output[blockOffset + bi] = temp[bi + bankOffsetB];
    }
}

/*!
    Step 2 of cumsummany:
    Compute cumulative sum on multiple arbitrary (small) portions of the subarrays on the GPU
 */
__global__ void prescan_many_arbitrary_kernel(double *output,
                                              double *input,
                                              unsigned long int dist,
                                              unsigned long int N,
                                              unsigned long int n,
                                              unsigned long int powerOfTwo)
{
    extern __shared__ double temp[];

    for (unsigned long int blockID = blockIdx.x; blockID < N; blockID += gridDim.x)
    {
        unsigned long int threadID = threadIdx.x;
        unsigned long int blockOffset = blockID * dist;

        // load input into shared memory
        unsigned long int ai = threadID;
        unsigned long int bi = threadID + (n / 2);
        unsigned long int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
        unsigned long int bankOffsetB = CONFLICT_FREE_OFFSET(bi);

        if (threadID < n)
        {
            temp[ai + bankOffsetA] = input[blockOffset + ai];
            temp[bi + bankOffsetB] = input[blockOffset + bi];
        }
        else
        {
            temp[ai + bankOffsetA] = 0.0;
            temp[bi + bankOffsetB] = 0.0;
        }

        // build sum in place up the tree
        unsigned long int offset = 1;
        for (unsigned long int d = powerOfTwo >> 1; d > 0; d >>= 1)
        {
            __syncthreads();
            if (threadID < d)
            {
                unsigned long int ai = offset * (2 * threadID + 1) - 1;
                unsigned long int bi = offset * (2 * threadID + 2) - 1;
                ai += CONFLICT_FREE_OFFSET(ai);
                bi += CONFLICT_FREE_OFFSET(bi);

                temp[bi] += temp[ai];
            }
            offset *= 2;
        }
        __syncthreads();

        // clear the last element
        if (threadID == 0)
        {
            temp[powerOfTwo - 1 + CONFLICT_FREE_OFFSET(powerOfTwo - 1)] = 0;
        }

        // traverse down tree and build scan
        for (int d = 1; d < powerOfTwo; d *= 2)
        {
            offset >>= 1;
            __syncthreads();
            if (threadID < d)
            {
                unsigned long int ai = offset * (2 * threadID + 1) - 1;
                unsigned long int bi = offset * (2 * threadID + 2) - 1;
                ai += CONFLICT_FREE_OFFSET(ai);
                bi += CONFLICT_FREE_OFFSET(bi);

                double t = temp[ai];
                temp[ai] = temp[bi];
                temp[bi] += t;
            }
        }
        __syncthreads();

        if (threadID < n)
        {
            output[blockOffset + ai] = temp[ai + bankOffsetA];
            output[blockOffset + bi] = temp[bi + bankOffsetB];
        }
    }
}

__global__ void add_many_kernel(double *output,
                                unsigned long int dist,
                                unsigned long int Nx,
                                unsigned long int NxNy,
                                unsigned long int n,
                                double *a)
{
    for (unsigned long int blockID = blockIdx.x; blockID < NxNy; blockID += gridDim.x)
    {
        unsigned long int threadID = threadIdx.x;
        if (threadID < n)
        {
            unsigned long int y = (blockID / Nx);
            unsigned long int blockOffset = y * dist + (blockID - y * Nx) * n;

            output[blockOffset + threadID] += a[blockID];
        }
    }
}

__global__ void add_many_kernel(double *output,
                                unsigned long int dist,
                                unsigned long int Nx,
                                unsigned long int NxNy,
                                unsigned long int n,
                                double *a1,
                                double *a2)
{
    for (unsigned long int blockID = blockIdx.x; blockID < NxNy; blockID += gridDim.x)
    {
        unsigned long int threadID = threadIdx.x;
        if (threadID < n)
        {
            unsigned long int y = (blockID / Nx);
            unsigned long int blockOffset = y * dist + (blockID - y * Nx) * n;

            output[blockOffset + threadID] += a1[blockID] + a2[blockID];
        }
    }
}

__global__ void copy_every_kernel(double *output,
                                  double *input,
                                  unsigned long int dist,
                                  unsigned long int N)
{
    for (unsigned long int blockID = blockIdx.x; blockID < N; blockID += gridDim.x)
    {
        output[blockID] = input[blockID * dist];
    }
}

/*!
    Compute next power of two larger or equal to n
 */
unsigned long int nextPowerOfTwo(unsigned long int n)
{
    unsigned long int power = 1;
    while (power < n)
    {
        power *= 2;
    }
    return power;
}