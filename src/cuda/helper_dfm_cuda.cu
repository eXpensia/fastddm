#include "hip/hip_runtime.h"
// Maintainer: enrico-lattuada

/*! \file helper_dfm_cuda.cu
    \brief Definition of helper functions for Digital Fourier Microscopy on the GPU
*/

// *** headers ***
#include "helper_dfm_cuda.cuh"
#include "helper_debug.cuh"

#include <hip/hip_runtime.h>

// *** code ***
const unsigned int TILE_DIM = 32;  // leave this unchanged!
const unsigned int BLOCK_ROWS = 8; // leave this unchanged!

/*!
    Convert array from float to double on device and prepare for fft2 (u_int8_t specialization)
*/
template <typename T>
__global__ void copy_convert_kernel(T *d_in,
                                    double *d_out,
                                    unsigned int width,
                                    unsigned int Npixels,
                                    unsigned int ipitch,
                                    unsigned int idist,
                                    unsigned int opitch,
                                    unsigned int odist,
                                    unsigned int N)
{
    for (unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < N; tid += blockDim.x * gridDim.x)
    {
        unsigned int t = tid / Npixels;
        unsigned int y = (tid - t * Npixels) / width;
        unsigned int x = tid - t * Npixels - y * width;

        T val = d_in[t * idist + y * ipitch + x];

        d_out[t * odist + y * opitch + x] = (double)val;
    }
}

template __global__ void copy_convert_kernel<double>(double *d_in, double *d_out, unsigned int width, unsigned int Npixels, unsigned int ipitch, unsigned int idist, unsigned int opitch, unsigned int odist, unsigned int N);
template __global__ void copy_convert_kernel<float>(float *d_in, double *d_out, unsigned int width, unsigned int Npixels, unsigned int ipitch, unsigned int idist, unsigned int opitch, unsigned int odist, unsigned int N);
template __global__ void copy_convert_kernel<int64_t>(int64_t *d_in, double *d_out, unsigned int width, unsigned int Npixels, unsigned int ipitch, unsigned int idist, unsigned int opitch, unsigned int odist, unsigned int N);
template __global__ void copy_convert_kernel<int32_t>(int32_t *d_in, double *d_out, unsigned int width, unsigned int Npixels, unsigned int ipitch, unsigned int idist, unsigned int opitch, unsigned int odist, unsigned int N);
template __global__ void copy_convert_kernel<int16_t>(int16_t *d_in, double *d_out, unsigned int width, unsigned int Npixels, unsigned int ipitch, unsigned int idist, unsigned int opitch, unsigned int odist, unsigned int N);
template __global__ void copy_convert_kernel<u_int64_t>(u_int64_t *d_in, double *d_out, unsigned int width, unsigned int Npixels, unsigned int ipitch, unsigned int idist, unsigned int opitch, unsigned int odist, unsigned int N);
template __global__ void copy_convert_kernel<u_int32_t>(u_int32_t *d_in, double *d_out, unsigned int width, unsigned int Npixels, unsigned int ipitch, unsigned int idist, unsigned int opitch, unsigned int odist, unsigned int N);
template __global__ void copy_convert_kernel<u_int16_t>(u_int16_t *d_in, double *d_out, unsigned int width, unsigned int Npixels, unsigned int ipitch, unsigned int idist, unsigned int opitch, unsigned int odist, unsigned int N);
template __global__ void copy_convert_kernel<u_int8_t>(u_int8_t *d_in, double *d_out, unsigned int width, unsigned int Npixels, unsigned int ipitch, unsigned int idist, unsigned int opitch, unsigned int odist, unsigned int N);

/*!
    Compute b = A * a
 */
__global__ void scale_array_kernel(double *a,
                                   double A,
                                   double *b,
                                   unsigned int N)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
    {
        b[i] = A * a[i];
    }
}