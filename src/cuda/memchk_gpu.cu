#include "hip/hip_runtime.h"
// Copyright (c) 2023-2023 University of Vienna, Enrico Lattuada, Fabian Krautgasser, and Roberto Cerbino.
// Part of FastDDM, released under the GNU GPL-3.0 License.

// Author: Enrico Lattuada
// Maintainer: Enrico Lattuada

/*! \file memchk_gpu.cu
    \brief Definition of utility functions for GPU memory check and optimization
*/

// *** headers ***
#include "memchk_gpu.cuh"
#include "gpu_utils.cuh"
#include "helper_cufft.cuh"
#include "helper_debug.cuh"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#include <iostream>

// *** code ***

/*!
    Evaluate the device memory pitch for multiple subarrays of size N with 16bytes elements
*/
void cudaGetDevicePitch16B(size_t N,
                           size_t &pitch)
{
    double2 *d_arr;

    gpuErrchk(hipMallocPitch(&d_arr, &pitch, N * sizeof(double2), 2));

    pitch /= sizeof(double2);

    gpuErrchk(hipFree(d_arr));
}

/*!
    Evaluate the device memory pitch for multiple subarrays of size N with 8bytes elements
*/
void cudaGetDevicePitch8B(size_t N,
                          size_t &pitch)
{
    double *d_arr;

    gpuErrchk(hipMallocPitch(&d_arr, &pitch, N * sizeof(double), 2));

    pitch /= sizeof(double);

    gpuErrchk(hipFree(d_arr));
}

/*!
    Evaluate the device memory pitch for multiple subarrays of size N with 4bytes elements
*/
void cudaGetDevicePitch4B(size_t N,
                          size_t &pitch)
{
    float *d_arr;

    gpuErrchk(hipMallocPitch(&d_arr, &pitch, N * sizeof(float), 2));

    pitch /= sizeof(float);

    gpuErrchk(hipFree(d_arr));
}

/*!
    Evaluate the device memory pitch for multiple subarrays of size N with 2bytes elements
*/
void cudaGetDevicePitch2B(size_t N,
                          size_t &pitch)
{
    int16_t *d_arr;

    gpuErrchk(hipMallocPitch(&d_arr, &pitch, N * sizeof(int16_t), 2));

    pitch /= sizeof(int16_t);

    gpuErrchk(hipFree(d_arr));
}

/*!
    Evaluate the device memory pitch for multiple subarrays of size N with 2bytes elements
*/
void cudaGetDevicePitch1B(size_t N,
                          size_t &pitch)
{
    int8_t *d_arr;

    gpuErrchk(hipMallocPitch(&d_arr, &pitch, N * sizeof(int8_t), 2));

    pitch /= sizeof(int8_t);

    gpuErrchk(hipFree(d_arr));
}

/*!
    Get device memory pitch (in number of elements)
*/
unsigned long long get_device_pitch(unsigned long long N,
                                    int num_bytes)
{
    size_t pitch;
    switch (num_bytes)
    {
    case 16:
        cudaGetDevicePitch16B(N, pitch);
        break;
    case 8:
        cudaGetDevicePitch8B(N, pitch);
        break;
    case 4:
        cudaGetDevicePitch4B(N, pitch);
        break;
    case 2:
        cudaGetDevicePitch2B(N, pitch);
        break;
    case 1:
        cudaGetDevicePitch1B(N, pitch);
        break;
    default:
        cudaGetDevicePitch8B(N, pitch);
    }

    return (unsigned long long)pitch;
}

/*!
    Optimize fft2 execution parameters based on available gpu memory.

    Writes in the corresponding arguments:
        - the number of iterations for fft2 (frame chunks)
        - the pitch in number of elements for buffer array (real values)

    Throws a runtime_error if the memory is not sufficient
    to perform the calculations.
*/
void optimize_fft2(unsigned long long width,
                   unsigned long long height,
                   unsigned long long length,
                   unsigned long long nx,
                   unsigned long long ny,
                   unsigned long long pixel_Nbytes,
                   bool is_input_Scalar,
                   bool is_window,
                   unsigned long long free_mem,
                   unsigned long long &pitch_buff,
                   unsigned long long &pitch_nx,
                   unsigned long long &num_fft2)
{
    /*
        Calculations are always performed in double precision.
        However, data is transferred as Scalar (float/double).

        To compute the fft2, we need (values are in bytes):
            - for the buffer (only if input is not Scalar):
                pitch_buff * height * num_fft2 * pixel_Nbytes
            - for the workspace (type: complex double [16 bytes]):
                (nx / 2 + 1) * ny * num_fft2 * 16
            - for the cufft2 internal buffer:
                [determined programmatically...]
            - for the window function (type: Scalar [SCALAR_SIZE bytes]):
                pitch_nx * height * 2 * SCALAR_SIZE
     */

    // Compute the effective number of grid points in x of the rfft2
    unsigned long long _nx = nx / 2ULL + 1ULL;

    // Get the pitch for the buffer array (only if the input is not Scalar)
    pitch_buff = is_input_Scalar ? 0ULL : get_device_pitch(width, pixel_Nbytes);

    // Get the pitch for the rfft2 output complex array
    pitch_nx = get_device_pitch(_nx, 2 * sizeof(Scalar));

    /*
        Start the optimization with the worst case scenario:
        we need to perform as many rfft2 loops as the number of images,
        namely, we transfer 1 image at a time.
     */
    num_fft2 = length;

    // Define auxiliary variables
    unsigned long long mem_required, prev_num_fft2;

    // Optimize
    while (true)
    {
        // Reset required memory value
        mem_required = 0;

        // Compute number of batched transforms
        unsigned long long batch = (length + num_fft2 - 1ULL) / num_fft2;

        // Estimate cufft2 internal work area size
        hipfftResult cufft_res;
        unsigned long long mem_fft2 = get_fft2_device_memory_size(nx,
                                                                  ny,
                                                                  batch,
                                                                  pitch_nx,
                                                                  cufft_res);

        // Check if the memory was retrieved successfully
        if (cufft_res == HIPFFT_SUCCESS)
        {
            // Add the required memory for cufft2
            mem_required += mem_fft2;

            // Add memory required for the work area
            mem_required += pitch_nx * ny * batch * 16ULL;

            // If input images do not have already the same type as the output,
            // add the required memory for the buffer array
            if (!is_input_Scalar)
            {
                mem_required += pitch_buff * height * batch * (unsigned long long)pixel_Nbytes;
            }

            // If user provided a window function,
            // add memory required for window
            if (is_window)
            {
                mem_required += pitch_nx * ny * 2 * sizeof(Scalar);
            }

            // Check memory and update parameters
            if (free_mem >= mem_required)
            {
                // Estimate the next numer of fft2 loops
                unsigned long long next_num_fft2 = (num_fft2 * mem_required + free_mem - 1ULL) / free_mem;

                // Check if the next number of fft2 loops is the same
                if (next_num_fft2 == prev_num_fft2)
                {
                    break;
                }
                else
                {
                    // Update and repeat
                    prev_num_fft2 = num_fft2;
                    num_fft2 = next_num_fft2;
                }
            }
            else if (num_fft2 == length)
            {
                // In this case, the available memory is less than the required
                // memory and the number of fft2 loops is already the maximum possible.
                // Therefore, we throw an error.
                throw std::runtime_error("Not enough space on GPU for fft2.");
            }
            else
            {
                num_fft2 = prev_num_fft2;
                break;
            }
        }
        else if (num_fft2 == length)
        {
            // The memory was not retrieved successfully
            // and the number of fft2 loops is already the maximum possible.
            // Therefore, we throw an error.
            throw std::runtime_error(
                "Not enough space on GPU for fft2. hipfftResult ID: " + cufft_res);
        }
    }
}

/*!
    Optimize structure function "diff" execution parameters based on available gpu memory
*/
void check_and_optimize_device_memory_diff(unsigned long long width,
                                           unsigned long long height,
                                           unsigned long long length,
                                           unsigned long long num_lags,
                                           unsigned long long nx,
                                           unsigned long long ny,
                                           int pixel_Nbytes,
                                           bool is_input_Scalar,
                                           bool is_window,
                                           unsigned long long &num_fft2,
                                           unsigned long long &num_chunks,
                                           unsigned long long &num_shift,
                                           unsigned long long &pitch_buff,
                                           unsigned long long &pitch_nx,
                                           unsigned long long &pitch_q,
                                           unsigned long long &pitch_t,
                                           unsigned long long &pitch_fs)
{
    // Get the available gpu memory
    unsigned long long free_mem = get_free_device_memory();

    // Scale the available memory by 0.9 to leave some free space
    free_mem = (unsigned long long)(0.9 * (double)free_mem);

    // Evaluate parameters for fft2
    optimize_fft2(width,
                  height,
                  length,
                  nx,
                  ny,
                  pixel_Nbytes,
                  is_input_Scalar,
                  is_window,
                  free_mem,
                  pitch_buff,
                  pitch_nx,
                  num_fft2);

    // Evaluate parameters for structure function ("diff" mode)

    // Evaluate parameters for fftshift
}
