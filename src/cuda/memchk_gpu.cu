#include "hip/hip_runtime.h"
// Copyright (c) 2023-2023 University of Vienna, Enrico Lattuada, Fabian Krautgasser, and Roberto Cerbino.
// Part of FastDDM, released under the GNU GPL-3.0 License.

// Author: Enrico Lattuada
// Maintainer: Enrico Lattuada

/*! \file memchk_gpu.cu
    \brief Definition of utility functions for GPU memory check and optimization
*/

// *** headers ***
#include "memchk_gpu.cuh"
#include "helper_debug.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <nvml.h>

// *** code ***

/*!
    Set the device to be used
*/
void PYBIND11_EXPORT set_device(int device_id)
{
    // Get number of available devices
    int deviceCount;
    hipError_t status = hipGetDeviceCount(&deviceCount);

    if (status != hipSuccess)
    {
        throw std::runtime_error("Failed to get CUDA device count. Error: " + std::string(hipGetErrorString(status)));
    }

    // Set device
    if (device_id < 0 || device_id >= deviceCount)
    {
        throw std::runtime_error("Invalid GPU ID provided. Valid ID range: 0 <= id < " + std::to_string(deviceCount));
    }
    else
    {
        int valid_devices[] = {device_id};
        hipSetValidDevices(valid_devices, 1);
    }
}

/*!
    Get free device memory (in bytes)
*/
unsigned long long PYBIND11_EXPORT get_free_device_memory()
{
    // Get set device
    int device_id;
    hipGetDevice(&device_id);

    // Get device available memory
    nvmlInit_v2();
    nvmlDevice_t dev;
    if (nvmlDeviceGetHandleByIndex_v2((unsigned int)device_id, &dev) != NVML_SUCCESS)
    {
        throw std::runtime_error("Failed to get device handle. Device ID: " + std::to_string(device_id));
    }
    nvmlMemory_t mem;
    nvmlDeviceGetMemoryInfo(dev, &mem);
    nvmlShutdown();

    return mem.free;
}

/*!
    Evaluate the device memory pitch for multiple subarrays of size N with 16bytes elements
*/
void cudaGetDevicePitch16B(size_t N,
                           size_t &pitch)
{
    double2 *d_arr;

    gpuErrchk(hipMallocPitch(&d_arr, &pitch, N * sizeof(double2), 2));

    pitch /= sizeof(double2);

    gpuErrchk(hipFree(d_arr));
}

/*!
    Evaluate the device memory pitch for multiple subarrays of size N with 8bytes elements
*/
void cudaGetDevicePitch8B(size_t N,
                          size_t &pitch)
{
    double *d_arr;

    gpuErrchk(hipMallocPitch(&d_arr, &pitch, N * sizeof(double), 2));

    pitch /= sizeof(double);

    gpuErrchk(hipFree(d_arr));
}

/*!
    Evaluate the device memory pitch for multiple subarrays of size N with 4bytes elements
*/
void cudaGetDevicePitch4B(size_t N,
                          size_t &pitch)
{
    float *d_arr;

    gpuErrchk(hipMallocPitch(&d_arr, &pitch, N * sizeof(float), 2));

    pitch /= sizeof(float);

    gpuErrchk(hipFree(d_arr));
}

/*!
    Evaluate the device memory pitch for multiple subarrays of size N with 2bytes elements
*/
void cudaGetDevicePitch2B(size_t N,
                          size_t &pitch)
{
    int16_t *d_arr;

    gpuErrchk(hipMallocPitch(&d_arr, &pitch, N * sizeof(int16_t), 2));

    pitch /= sizeof(int16_t);

    gpuErrchk(hipFree(d_arr));
}

/*!
    Evaluate the device memory pitch for multiple subarrays of size N with 2bytes elements
*/
void cudaGetDevicePitch1B(size_t N,
                          size_t &pitch)
{
    int8_t *d_arr;

    gpuErrchk(hipMallocPitch(&d_arr, &pitch, N * sizeof(int8_t), 2));

    pitch /= sizeof(int8_t);

    gpuErrchk(hipFree(d_arr));
}

/*!
    Get device memory pitch (in number of elements)
*/
unsigned long long get_device_pitch(unsigned long long N,
                                    int num_bytes)
{
    size_t pitch;
    switch (num_bytes)
    {
    case 16:
        cudaGetDevicePitch16B(N, pitch);
        break;
    case 8:
        cudaGetDevicePitch8B(N, pitch);
        break;
    case 4:
        cudaGetDevicePitch4B(N, pitch);
        break;
    case 2:
        cudaGetDevicePitch2B(N, pitch);
        break;
    case 1:
        cudaGetDevicePitch1B(N, pitch);
        break;
    default:
        cudaGetDevicePitch8B(N, pitch);
    }

    return (unsigned long long)pitch;
}

/*!
    Optimize fft2 execution parameters based on available gpu memory.

    Writes in the corresponding arguments:
        - the number of iterations for fft2 (frame chunks)
        - the pitch in number of elements for buffer array (real values)

    Throws a runtime_error if the memory is not sufficient
    to perform the calculations.
*/
void optimize_fft2(unsigned long long width,
                   unsigned long long height,
                   unsigned long long length,
                   unsigned long long nx,
                   unsigned long long ny,
                   unsigned long long pixel_Nbytes,
                   bool is_input_Scalar,
                   bool is_window,
                   unsigned long long free_mem,
                   unsigned long long &pitch_buff,
                   unsigned long long &pitch_nx,
                   unsigned long long &num_fft2)
{
    /*
        Calculations are always performed in double precision.
        However, data is transferred as Scalar (float/double).

        To compute the fft2, we need (values are in bytes):
            - for the buffer (only if input is not Scalar):
                pitch_buff * height * num_fft2 * pixel_Nbytes
            - for the workspace (type: complex double [16 bytes]):
                (nx / 2 + 1) * ny * num_fft2 * 16
            - for the cufft2 internal buffer:
                [determined programmatically...]
            - for the window function (type: Scalar [SCALAR_SIZE bytes]):
                pitch_nx * height * 2 * SCALAR_SIZE
     */

    // Compute the effective number of grid points in x of the rfft2
    unsigned long long _nx = nx / 2ULL + 1ULL;

    // Get the pitch for the buffer array (only if the input is not Scalar)
    pitch_buff = is_input_Scalar ? 0ULL : get_device_pitch(width, pixel_Nbytes);

    // Get the pitch for the rfft2 output complex array
    pitch_nx = get_device_pitch(_nx, 2 * sizeof(Scalar));

    /*
        Start the optimization with the worst case scenario:
        we need to perform as many rfft2 loops as the number of images,
        namely, we transfer 1 image at a time.
     */
    num_fft2 = length;

    // Define auxiliary variables
    unsigned long long mem_required, prev_num_fft2;

    // Optimize
}

/*!
    Optimize structure function "diff" execution parameters based on available gpu memory
*/
void check_and_optimize_device_memory_diff(unsigned long long width,
                                           unsigned long long height,
                                           unsigned long long length,
                                           unsigned long long num_lags,
                                           unsigned long long nx,
                                           unsigned long long ny,
                                           int pixel_Nbytes,
                                           bool is_input_Scalar,
                                           bool is_window,
                                           unsigned long long &num_fft2,
                                           unsigned long long &num_chunks,
                                           unsigned long long &num_shift,
                                           unsigned long long &pitch_buff,
                                           unsigned long long &pitch_nx,
                                           unsigned long long &pitch_q,
                                           unsigned long long &pitch_t,
                                           unsigned long long &pitch_fs)
{
    // Get the available gpu memory
    unsigned long long free_mem = get_free_device_memory();

    // Scale the available memory by 0.9 to leave some free space
    free_mem = (unsigned long long)(0.9 * (double)free_mem);

    // Evaluate parameters for fft2
    optimize_fft2(width,
                  height,
                  length,
                  nx,
                  ny,
                  pixel_Nbytes,
                  is_input_Scalar,
                  is_window,
                  free_mem,
                  pitch_buff,
                  pitch_nx,
                  num_fft2);

    // Evaluate parameters for structure function ("diff" mode)

    // Evaluate parameters for fftshift
}
